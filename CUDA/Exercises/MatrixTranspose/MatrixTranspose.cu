#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
    /// YOUR CODE
}

const int N  = 1000;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    /// SAFE_CALL( hipMalloc( ... ) )
    /// SAFE_CALL( hipMalloc( ... ) )

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    /// SAFE_CALL( hipMemcpy( ... ) )

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    /// matrixTransposeKernel<<< , >>>();

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    /// SAFE_CALL( hipMemcpy( ... ) )

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    /// SAFE_CALL( hipFree( ... ) )
    /// SAFE_CALL( hipFree( ... ) )

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
