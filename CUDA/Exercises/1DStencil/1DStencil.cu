#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int RADIUS = 7;

__global__
void stencilKernel(const int* d_input, int N,int* d_output) {
    // YOUR CODE
    
    
    int idThreads = blockIdx.x * blockDim.x * threadIdx.x;
    
    int valueArr = 0;
    
    if (idThreads > RADIUS-1 && idThread < N - RADIUS)
    {
        for (int i = 1; i <= RADIUS; i++)
        {
            valueArr += d_input[idThreads+i];
            valueArr += d_input[idThreads-i];
        }

        d_output = valueArr + d_input[idThreads];
    }
    
}

const int N  = 1000;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_input      = new int[N];
    int* h_output_tmp = new int[N]; // <-- used for device result
    int* h_output     = new int[N](); // initilization to zero

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N; i++)
        h_input[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = RADIUS; i < N - RADIUS; i++) {
        for (int j = i - RADIUS; j <= i + RADIUS; j++)
            h_output[i] += h_input[j];
    }

    TM_host.stop();
    TM_host.print("1DStencil host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_input, *d_output;
    SAFE_CALL( hipMalloc(&d_input, N * sizeof(int)));
    SAFE_CALL( hipMalloc(&d_output, N * sizeof(int)));

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL(hipMemcpy(d_input, h_input, N * sizeof(int),hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // did you miss something?
    ///
    
    dim3 block_size(128,1,1);
    dim3 num_blocks(N/128,1,1);
    if(N%128)
    {
        block_size.x++;
    }

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();

    stencilKernel<<< num_blocks, block_size>>>(d_input, N ,d_output);

    CHECK_CUDA_ERROR
    TM_device.stop();
    
    TM_device.print("1DStencil device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL(hipMemcpy(h_output, d_output, N * sizeof(int),hipMemcpyDeviceToHost));

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N; i++) {
        if (h_output[i] != h_output_tmp[i]) {
            std::cerr << "wrong result at: " << i
                      << "\nhost:   " << h_output[i]
                      << "\ndevice: " << h_output_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_input;
    delete[] h_output;
    delete[] h_output_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    /// SAFE_CALL( hipFree( ... ) )
    /// SAFE_CALL( hipFree( ... ) )

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
